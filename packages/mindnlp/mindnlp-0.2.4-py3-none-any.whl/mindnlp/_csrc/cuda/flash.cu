// Modified from: https://github.com/tspeterkim/flash-attention-minimal/blob/main/flash.cu  
#include <stdio.h>
#include <hip/hip_runtime.h>


#define ENABLE_NOTE_LOG 0

__global__ void flash_attn_2_fwd_f32_kernel(
  const float* Q, 
  const float* K, 
  const float* V, 
  const int N, 
  const int d,
  const int Tc,
  const int Tr, 
  const int Bc, 
  const int Br, 
  const float softmax_scale,
  float* l, 
  float *m, 
  float* O) {
  int tx = threadIdx.x;
  int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

  // Offset into Q,K,V,O,l,m - different for each batch and head
  int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
  int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for l and m

  // Define SRAM for Q,K,V,S
  extern __shared__ float sram[];
  int tile_size = Bc * d;  // size of Qi, Kj, Vj
  float* Qi = sram;
  float* Kj = &sram[tile_size];
  float* Vj = &sram[tile_size * 2];
  float* S = &sram[tile_size * 3];
  
  // TODO: swap the load order of Kj, Vj and Qi. first load Qi, then Kj, Vj
  for (int j = 0; j < Tc; j++) {

      // Load Kj, Vj to SRAM
      for (int x = 0; x < d; x++) {
          Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
          Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
      }
      __syncthreads();  // such that the inner loop can use the correct Kj, Vj

      for (int i = 0; i < Tr; i++)  {

          // Load Qi to SRAM, l and m to registers
          for (int x = 0; x < d; x++) {
              Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
          }
          float row_m_prev = m[lm_offset + (Br * i) + tx];
          float row_l_prev = l[lm_offset + (Br * i) + tx];

          // S = QK^T, row_m = rowmax(S)
          float row_m = -INFINITY;
          for (int y = 0; y < Bc; y++) {
              float sum = 0;
              for (int x = 0; x < d; x++) {
                  sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
              }
              sum *= softmax_scale;
              S[(Bc * tx) + y] = sum;

              if (sum > row_m)
                  row_m = sum;
          }

          // P = exp(S - row_m), row_l = rowsum(P)
          float row_l = 0;
          for (int y = 0; y < Bc; y++) {
              S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m);
              row_l += S[(Bc * tx) + y];
          }

          // Compute new m and l
          float row_m_new = max(row_m_prev, row_m);
          float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

          // Write O, l, m to HBM
          for (int x = 0; x < d; x++) {
              float pv = 0;  // Pij * Vj
              for (int y = 0; y < Bc; y++) {
                  pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
              }
              O[qkv_offset + (tile_size * i) + (tx * d) + x] = (1 / row_l_new) \
                  * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size * i) + (tx * d) + x]) \
                  + (__expf(row_m - row_m_new) * pv));
          }
          m[lm_offset + (Br * i) + tx] = row_m_new;
          l[lm_offset + (Br * i) + tx] = row_l_new;
      }
      __syncthreads();  // otherwise, thread can use the wrong Kj, Vj in inner loop
  }
}


extern "C" {

    int flash_forward(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes, void *stream,
                    void *extra) {
        hipStream_t custream = static_cast<hipStream_t>(stream);
        if (nparam != 6) return 1;
        float *Q = static_cast<float *>(params[0]);
        float *K = static_cast<float *>(params[1]);
        float *V = static_cast<float *>(params[2]);
        // put l,m as input params
        float *l = static_cast<float *>(params[3]);
        float *m = static_cast<float *>(params[4]);
        float *O = static_cast<float *>(params[5]);

        const int B = static_cast<int>(shapes[0][0]);
        const int nh = static_cast<int>(shapes[0][1]);
        const int N = static_cast<int>(shapes[0][2]);
        const int d = static_cast<int>(shapes[0][3]);

        int max_sram_size;
        hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

        // set block size, TODO: dynamically set block size
        const int Bc = 32;
        const int Br = 32;

        const int Tc = ceil((float) N / Bc);
        const int Tr = ceil((float) N / Br);
        const float softmax_scale = 1.0 / sqrt(d);

        // Calculate SRAM size needed per block
        const int sram_size = (2 * Bc * d * sizeof(float)) + (4 * Br * d * sizeof(float));
        printf("Bc: %d, Br: %d, Tc: %d, Tr: %d \n", Bc, Br, Tc, Tr);
        printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

        dim3 grid_dim(B, nh);  // batch_size x num_heads
        dim3 block_dim(Bc);  // Bc threads per block

        flash_attn_2_fwd_f32_kernel<<<grid_dim, block_dim, sram_size, custream>>>(
            Q, K, V, N, d, Tc, Tr, Bc, Br, softmax_scale, l, m, O
        );
        return 0;
    }
}